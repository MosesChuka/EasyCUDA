#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    kernel<<<1, 4>>>();
    hipDeviceSynchronize();
    return 0;
}